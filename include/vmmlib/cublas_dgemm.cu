#ifndef __VMML__VMMLIB_CUBLAS_DGEMM__HPP__
#define __VMML__VMMLIB_CUBLAS_DGEMM__HPP__


#include <vmmlib/matrix.hpp>
#include <vmmlib/exception.hpp>
#include <vmmlib/cublas_includes.hpp>
#include <vmmlib/cublas_types.hpp>

/** 
 *
 *   a wrapper for CUBLAS DGEMM routine. 
 
 hipblasStatus_t hipblasDgemm(
			hipblasHandle_t handle,
			hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k,
			const double const double const double const double double
			*alpha, *A, int lda, *B, int ldb, *beta,
			*C, int ldc
			)
 
 *
 *  Purpose
 *  =======
 *
 *  CUBLAS DGEMM  is a CUDA implementation of the level3 BLAS DGEMM implementation
 *  performs one of the matrix-matrix operations
 *
 *     C := alpha*op( A )*op( B ) + beta*C,
 *
 *  where  op( X ) is one of
 *
 *     op( X ) = X   or   op( X ) = X**T,
 *
 *  alpha and beta are scalars, and A, B and C are matrices, with op( A )
 *  an m by k matrix,  op( B )  a  k by n matrix and  C an m by n matrix.
 *
 *
 *   more information in: http://www.netlib.org/blas/dgemm.f 
 *   or http://www.netlib.org/clapack/cblas/dgemm.c
 **
 */


namespace vmml
{
	
	namespace cublas
	{
		
#if 0
		/* CUBLAS DGEMM Subroutine */ 
		hipblasStatus_t hipblasDgemm(
								   hipblasHandle_t handle,
								   hipblasOperation_t transa, hipblasOperation_t transb, 
								   int m, int n, int k,
								   const double *alpha, 
								   const double *A, int lda,
								   const double *B, int ldb,
								   const *beta,
								   double *C, int ldc
								   )

		/* DGEMM Subroutine */ 
		void cblas_dgemm(enum CBLAS_ORDER Order, enum CBLAS_TRANSPOSE TransA, enum CBLAS_TRANSPOSE TransB, 
						 blasint M, blasint N, blasint K,
						 double alpha, double *A, blasint lda, double *B, blasint ldb, double beta, double *C, blasint ldc);
		
#endif
		
		template< typename float_t >
		struct dgemm_params
		{
			//FIXME maybe add const 
			hipblasHandle_t		handle;
			hipblasOperation_t	trans_a;
			hipblasOperation_t	trans_b;
			cublas_int			m;
			cublas_int			n;
			cublas_int			k;
			float_t				alpha;
			float_t*			h_a; //host
			float_t*			d_a; //device
			cublas_int			lda; //leading dimension of input array matrix left
			float_t*			h_b;
			float_t*			d_b;
			cublas_int			ldb; //leading dimension of input array matrix right
			float_t				beta;
			float_t*			h_c;
			float_t*			d_c;
			cublas_int			ldc; //leading dimension of output array matrix right
			
			friend std::ostream& operator << ( std::ostream& os, 
											  const dgemm_params< float_t >& p )
			{
				os 
				<< " (1)\thandle "		<< p.handle << std::endl
				<< " (2)\ttrans_a "		<< p.trans_a << std::endl
				<< " (3)\ttrans_b "     << p.trans_b << std::endl
				<< " (4)\tm "			<< p.m << std::endl
				<< " (6)\tn "			<< p.n << std::endl
				<< " (5)\tk "			<< p.k << std::endl
				<< " (7)\talpha "       << p.alpha << std::endl 
				<< " (8)\th_a "			<< p.h_a << std::endl
				<< " (9)\tlda "			<< p.lda << std::endl
				<< " (10)\th_b "		<< p.h_b << std::endl
				<< " (11)\tldb "		<< p.ldb << std::endl
				<< " (12)\tbeta "       << p.beta << std::endl
				<< " (13)\th_c "		<< p.h_c << std::endl
				<< " (14)\tldc "        << p.ldc << std::endl
				<< std::endl;
				return os;
			}
			
		};
		
		
		
		template< typename float_t >
		inline void
		dgemm_call( dgemm_params< float_t >& p )
		{
			VMMLIB_ERROR( "not implemented for this type.", VMMLIB_HERE );
		}
		
		
		template<>
		inline void
		dgemm_call( dgemm_params< float >& p )
		{
			//std::cout << "calling cublas sgemm (single precision) " << std::endl;
			hipblasStatus_t stat  = hipblasSgemm( 
						p.handle,
						p.trans_a,
						p.trans_b,
						p.m,
						p.n,
						p.k,
						&p.alpha,
						p.d_a,
						p.lda,
						p.d_b,
						p.ldb,
						&p.beta,
						p.d_c,
						p.ldc
						);
			
		}
		
		template<>
		inline void
		dgemm_call( dgemm_params< double >& p )
		{
			//std::cout << "calling cublas dgemm (double precision) " << std::endl;

			hipblasStatus_t stat  = hipblasDgemm( 
						p.handle,
						p.trans_a,
						p.trans_b,
						p.m,
						p.n,
						p.k,
						&p.alpha,
						p.d_a,
						p.lda,
						p.d_b,
						p.ldb,
						&p.beta,
						p.d_c,
						p.ldc
						);
		}
		
	} // namespace cublas
	
	
	
	template< size_t M, size_t K, size_t N, typename float_t >
	struct cublas_dgemm
	{
		
		typedef matrix< M, K, float_t > matrix_left_t;
		typedef matrix< K, M, float_t > matrix_left_t_t;
		typedef matrix< K, N, float_t > matrix_right_t;
		typedef matrix< N, K, float_t > matrix_right_t_t;
		typedef matrix< M, N, float_t > matrix_out_t;
		typedef vector< M, float_t > vector_left_t;
		typedef vector< N, float_t > vector_right_t;
		
		cublas_dgemm();
		~cublas_dgemm();
		
		bool compute( const matrix_left_t& A_, const matrix_right_t& B_, matrix_out_t& C_ );
		bool compute( const matrix_left_t& A_, matrix_out_t& C_ );
		
		
		cublas::dgemm_params< float_t > p;
		
		const cublas::dgemm_params< float_t >& get_params(){ return p; };
		
		
	}; // struct cublas_dgemm
	
	
	template< size_t M, size_t K, size_t N, typename float_t >
	cublas_dgemm< M, K, N, float_t >::cublas_dgemm()
	{
        hipblasStatus_t cstat = hipblasCreate( &p.handle ); if ( cstat > 0 ) { printf( "hipblasCreate: status error=%d\n", cstat ); }
		p.trans_a    = HIPBLAS_OP_N;
		p.trans_b    = HIPBLAS_OP_N;
		p.m          = M;
		p.n          = N;
		p.k          = K;
		p.alpha      = 1.0f;
		p.h_a        = 0;
		p.d_a        = 0;
		p.lda        = M;
		p.h_b        = 0;
		p.d_b        = 0;
		p.ldb        = K; //no transpose, use N for transpose
		p.beta       = 0.0;
		p.h_c        = 0;
		p.d_c        = 0;
		p.ldc        = M;
	}
	
	template< size_t M, size_t K, size_t N, typename float_t >
	cublas_dgemm< M, K, N, float_t >::~cublas_dgemm()
	{
		/*hipblasStatus_t cuerr = hipblasDestroy( p.handle );
		if ( cuerr > 0 ) 
		{ printf( "hipMemcpy: cublas error=%d\n", cuerr ); }*/
	}

	
	
	
	template< size_t M, size_t K, size_t N, typename float_t >
	bool
	cublas_dgemm< M, K, N, float_t >::compute( 
											const matrix_left_t& A_, 
											const matrix_right_t& B_,
											matrix_out_t& C_ 
											)
	{
		// cublas needs non-const data
		matrix_left_t* AA = new matrix_left_t( A_ );
		matrix_right_t* BB = new matrix_right_t( B_ );
		C_.zero();
		
		p.h_a         = AA->array;
		p.h_b         = BB->array;
		p.h_c         = C_.array;
				
		// memory sizes of matrices
		size_t mem_size_A = sizeof(float_t) * M * K;
		size_t mem_size_B = sizeof(float_t) * K * N;
		size_t mem_size_C = sizeof(float_t) * M * N;
		
		// allocate device memory
		hipError_t cuerr = hipMalloc( (void**) &p.d_a, mem_size_A ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); }
		cuerr = hipMalloc( (void**) &p.d_b, mem_size_B ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); } 
		cuerr = hipMalloc( (void**) &p.d_c, mem_size_C ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); }  
		
		// copy host memory to device
		cuerr = hipMemcpy( p.d_a, p.h_a, mem_size_A, hipMemcpyHostToDevice); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }
		cuerr = hipMemcpy( p.d_b, p.h_b, mem_size_B, hipMemcpyHostToDevice); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }

		// call CUBLAS V2
		cublas::dgemm_call< float_t >( p );
		//std::cout << p << std::endl; //debug
	
		hipDeviceSynchronize();

		// copy result from device to host
		cuerr = hipMemcpy( p.h_c, p.d_c, mem_size_C, hipMemcpyDeviceToHost); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }
		
		// clean up memory
		hipFree( p.d_a );
		hipFree( p.d_b );
		hipFree( p.d_c );
		hipDeviceReset();
		
		delete AA;
		delete BB;
		
		return true;
	}	
	
	
	template< size_t M, size_t K, size_t N, typename float_t >
	bool
	cublas_dgemm< M, K, N, float_t >::compute( 
											const matrix_left_t& A_, 
											matrix_out_t& C_ 
											)
	{
		// cublas needs non-const data
		matrix_left_t* AA = new matrix_left_t( A_ );
		C_.zero();
		
		p.h_a         = AA->array;
		p.h_b         = AA->array;
		p.h_c         = C_.array;
		p.trans_b     = HIPBLAS_OP_T;
		p.ldb         = N; 
		
		// memory sizes of matrices
		size_t mem_size_A = sizeof(float_t) * M * K;
		size_t mem_size_B = sizeof(float_t) * K * N;
		size_t mem_size_C = sizeof(float_t) * M * N;
		
		// allocate device memory
		hipError_t cuerr = hipMalloc( (void**) &p.d_a, mem_size_A ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); }
		cuerr = hipMalloc( (void**) &p.d_b, mem_size_B ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); } 
		cuerr = hipMalloc( (void**) &p.d_c, mem_size_C ); if ( cuerr > 0 ) { printf( "hipMalloc: cublas error=%d\n", cuerr ); }  
		
		// copy host memory to device
		cuerr = hipMemcpy( p.d_a, p.h_a, mem_size_A, hipMemcpyHostToDevice); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }
		cuerr = hipMemcpy( p.d_b, p.h_b, mem_size_B, hipMemcpyHostToDevice); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }

		// call CUBLAS V2
		cublas::dgemm_call< float_t >( p );
		//std::cout << p << std::endl; //debug
	
		hipDeviceSynchronize();

		// copy result from device to host
		cuerr = hipMemcpy( p.h_c, p.d_c, mem_size_C, hipMemcpyDeviceToHost); if ( cuerr > 0 ) { printf( "hipMemcpy: cublas error=%d\n", cuerr ); }
		
		// clean up memory
		hipFree( p.d_a );
		hipFree( p.d_b );
		hipFree( p.d_c );
		hipDeviceReset();

		delete AA;
		
		return true;
	}	

	
	
} // namespace vmml

#endif	

